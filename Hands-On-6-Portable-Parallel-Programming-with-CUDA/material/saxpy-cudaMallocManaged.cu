#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void saxpy(int n, float * x, float * y) {
  int i = threadIdx.x;
  if (i < n)
    y[i] = x[i] + y[i];
}

void printVector(float * vector, int n) {
  for (int i = 0; i < n; ++i)
    printf("%1.0f\t", vector[i]);
  printf("\n\n");
}

void generateVector(float * vector, int n) {
  for (int i = 0; i < n; ++i)
    vector[i] = i + 1;
}

int main(int argc, char * argv[]) {
  int n = atoi(argv[1]);
  float * x, * y;

  hipMallocManaged( & x, sizeof(float) * n);
  hipMallocManaged( & y, sizeof(float) * n);

  generateVector(x, n);
  printVector(x, n);

  generateVector(y, n);
  printVector(y, n);

  int NUMBER_OF_BLOCKS = 1;
  int NUMBER_OF_THREADS_PER_BLOCK = n;

  saxpy << < NUMBER_OF_BLOCKS, NUMBER_OF_THREADS_PER_BLOCK >>> (n, x, y);

  hipDeviceSynchronize();

  printVector(y, n);

  hipFree(x);
  hipFree(y);
  
  return 0;
}