/****************************************************************************80
*  Code: 
*   saxpy-gpu.cu
*
*  Purpose:
*   Implements in C the simple SAXPY stands for Single-Precision with GPU.
*
*  Modified:
*   May 08 2022 17:28 
*
*  Author:
*    Murilo Boratto  < muriloboratto 'at' fieb.org.br >
*
*  HowtoCompile:
*    gcc saxpy-gpu.cu -o saxpy-gpu
*   
*  HowtoExecute:
*    ./saxpy-gpu <size>
*    ./saxpy-gpu  10
*
*******************************************************************************/

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void saxpy(int n, float *x, float *y) {
  int i = threadIdx.x;
  if(i < n)
  y[i] = x[i] + y[i];
}

void printVector(float *vector, int n){

for (int i=0; i < n ; ++i)
 printf("%1.0f\t", vector[i]);

 printf("\n\n");

}

void generateVector(float *vector, int n){

for (int i=0; i < n ; ++i)
 vector[i] = i + 1;

}

int main(int argc, char *argv[]){

  int n = atoi(argv[1]);   
  float *x,*y;
  float *xd, *yd;

  x = (float*) malloc(sizeof(float) * n);
  y = (float*) malloc(sizeof(float) * n);

  hipMalloc( (void**)&xd, sizeof(float) * n );
  hipMalloc( (void**)&yd, sizeof(float) * n );
 
  generateVector(x, n);
  printVector(x, n);

  generateVector(y, n);
  printVector(y, n);

  hipMemcpy(xd, x, sizeof(float) * n, hipMemcpyHostToDevice);
  hipMemcpy(yd, y, sizeof(float) * n, hipMemcpyHostToDevice);

  int NUMBER_OF_BLOCKS = 1;
  int NUMBER_OF_THREADS_PER_BLOCK = n;

  saxpy<<< NUMBER_OF_BLOCKS, NUMBER_OF_THREADS_PER_BLOCK >>>(n, xd, yd);

  hipDeviceSynchronize();

  hipMemcpy(y, yd, sizeof(float) * (n), hipMemcpyDeviceToHost);

  printVector(y, n);
 
  free(x);
  free(y);

  hipFree(xd);
  hipFree(yd);

  return 0;

}

