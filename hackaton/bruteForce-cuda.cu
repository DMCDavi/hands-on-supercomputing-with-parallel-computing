#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <math.h>
#include <hip/hip_runtime.h>

//97 to 122 use only lowercase letters
//65 to 90 use only capital letters
//48 to 57 use only numbers

#define START_CHAR 97
#define END_CHAR 122
#define MAXIMUM_PASSWORD 20
#define MAX_THREADS_PER_BLOCK 1024

long long my_pow(long long x, int y)
{
  long long res = 1;
  if (y==0)
    return res;
  else
    return x * my_pow(x, y-1);
}

__global__ void bruteForce_LOOP(char *sc, long long int max, long long int pass_decimal, int base){
  long long int j = threadIdx.x * blockIdx.x;
  
  if(j<max){
    
    if(j == pass_decimal){
      int index = 0;

      while(j > 0){
        sc[index++] = 'a' + j%base-1;
        j /= base;
      }
      sc[index] = '\0';
      
      return;
   
    }
}

}


void bruteForce(char *pass) 
{
  char force[MAXIMUM_PASSWORD];
  int palavra[MAXIMUM_PASSWORD];
  int pass_b26[MAXIMUM_PASSWORD];

  
  long long int pass_decimal = 0;
  int base = END_CHAR - START_CHAR + 2;

  int size = strlen(pass);

  for(int i = 0; i < MAXIMUM_PASSWORD; i++)
    force[i] = '\0';

  printf("Try to broke the password: %s\n", pass);

  for(int i = 0; i < size; i++)
    pass_b26[i] = (int) pass[i] - START_CHAR + 1; 

  for(int i = size - 1; i > -1; i--)
    pass_decimal += (long long int) pass_b26[i] * my_pow(base, i);

  long long int max = my_pow(base, size);
 

  int NUMBER_OF_BLOCKS = max/MAX_THREADS_PER_BLOCK + 1;
  int NUMBER_OF_THREADS_PER_BLOCK = MAX_THREADS_PER_BLOCK;
printf("%d\n", NUMBER_OF_BLOCKS);
printf("%d\n", NUMBER_OF_THREADS_PER_BLOCK);
printf("%lld\n", max);
printf("%lld\n", pass_decimal);
printf("%d\n", base);

char *sc;
  hipMallocManaged( & sc, sizeof(char) * MAXIMUM_PASSWORD);

  bruteForce_LOOP<<<NUMBER_OF_BLOCKS, NUMBER_OF_THREADS_PER_BLOCK>>>(sc, max, pass_decimal, base);
  hipDeviceSynchronize();
  printf("Found password!\n");
  printf("Found password: %s\n", sc);

  hipFree(sc);
}

int main(int argc, char **argv) 
{
  char password[MAXIMUM_PASSWORD];

  strcpy(password, argv[1]);
  time_t t1, t2;
  double dif;

  
  time (&t1);
    bruteForce(password);
  time (&t2);

  dif = difftime (t2, t1);

  printf("\n%1.2f seconds\n", dif);

  return 0;
}