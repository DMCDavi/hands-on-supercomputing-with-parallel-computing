#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <assert.h>

//97 to 122 use only lowercase letters
//65 to 90 use only capital letters
//48 to 57 use only numbers

#define START_CHAR 97
#define END_CHAR 122
#define MAXIMUM_PASSWORD 20
#define MAX_THREADS_PER_BLOCK 1024
#define BLOCKS_PER_SM 32

// Retorna erro, se houver, de uma função CUDA
inline hipError_t checkCuda(hipError_t result)
{
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    assert(result == hipSuccess);
  }
  return result;
}

__device__ long long my_pow(long long x, int y) {
  long long res = 1;
  if (y == 0)
    return res;
  else
    return x * my_pow(x, y - 1);
}

// Calcula o tamanho de uma string
// Obs.: CUDA não tem suporte para a função strlen de string.h
__device__ int my_strlen(char *s) {
  int sum = 0;
  while (*s++) sum++;
  return sum;
}

__global__ void bruteForce(char * pass) {
  int pass_b26[MAXIMUM_PASSWORD];

  long long int pass_decimal = 0;
  int base = END_CHAR - START_CHAR + 2;

  int size = my_strlen(pass);;

  for (int i = 0; i < size; i++)
    pass_b26[i] = (int) pass[i] - START_CHAR + 1;

  for (int i = size - 1; i > -1; i--)
    pass_decimal += (long long int) pass_b26[i] * my_pow(base, i);

  long long int max = my_pow(base, size);
  char s[MAXIMUM_PASSWORD];
  // Calcula a iteração a partir da quantidade de threads vezes o id do bloco mais o id da thread
  long long int j = blockIdx.x * blockDim.x + threadIdx.x;

  // Realiza loop-stride para processar dados maiores que a quantidade de threads na GPU
  // Uma única thread processa mais de um dado
  while (j < max) {
    if (j == pass_decimal) {
      // printf("Found password!\n");
      int index = 0;

      // printf("Password in decimal base: %lli\n", j);
      while (j > 0) {
        s[index++] = 'a' + j % base - 1;
        j /= base;
      }
      s[index] = '\0';

      // printf("Found password: %s\n", s);
      break;
    }
    // Calcula o stride pela multiplicação entre a quantidade de blocos e threads 
    j += blockDim.x * gridDim.x;
  }
}

int main(int argc, char ** argv) {
  char *password;
  time_t t1, t2;
  double dif;

  // Aloca a senha para ser acessada tanto pela CPU quanto pela GPU
  checkCuda( hipMallocManaged( & password, MAXIMUM_PASSWORD * sizeof(char)) );

  strcpy(password, argv[1]);

  int deviceId, numberOfSMs;
  checkCuda( hipGetDevice( & deviceId) );
  // Pega a quantidade de SMs presentes na GPU
  checkCuda( hipDeviceGetAttribute( & numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId) );
  // Multiplica a quantidade de SMs pela quantidade de blocos presentes em cada um
  int number_of_blocks = numberOfSMs * BLOCKS_PER_SM;
  int threads_per_block = MAX_THREADS_PER_BLOCK;

  // printf("Try to broke the password: %s\n", password);

  time( & t1);
  bruteForce <<< number_of_blocks, threads_per_block >>> (password);
  checkCuda( hipGetLastError() );
  checkCuda( hipDeviceSynchronize() );
  time( & t2);

  dif = difftime(t2, t1);
  printf("B%dT%d;%1.2f\n", number_of_blocks, threads_per_block, dif);

  checkCuda( hipFree(password) );

  return 0;
}